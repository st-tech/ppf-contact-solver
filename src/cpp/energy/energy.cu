#include "hip/hip_runtime.h"
// File: energy.cu
// Author: Ryoichi Ando (ryoichi.ando@zozo.com)
// License: Apache v2.0


#include "../eigenanalysis/eigenanalysis.hpp"
#include "../utility/dispatcher.hpp"
#include "../utility/utility.hpp"
#include "model/air_damper.hpp"
#include "model/arap.hpp"
#include "model/baraffwitkin.hpp"
#include "model/dihedral_angle.hpp"
#include "model/hook.hpp"
#include "model/momentum.hpp"
#include "model/snhk.hpp"
#include "model/stvk.hpp"

namespace energy {

__device__ void embed_vertex_force_hessian(
    const DataSet &data, const Vec<Vec3f> &eval_x, const Vec<Vec3f> &velocity,
    const Vec<Vec3f> &target, Vec<float> &force, Vec<Mat3x3f> &diag_hess,
    float dt, const ParamSet &param, unsigned i) {

    float mass = data.prop.vertex[i].mass;
    float area = data.prop.vertex[i].area;

    const Vec3f &x = data.vertex.curr[i];
    const Vec3f &y = eval_x[i];
    const Vec3f normal = utility::compute_vertex_normal(data, eval_x, i);

    Vec3f wind = Vec3f::Zero();
    if (!param.fitting) {
        wind = utility::get_wind_weight(param.time) * param.wind;
    }

    Vec3f f = Vec3f::Zero();
    Mat3x3f H = Mat3x3f::Zero();
    if (normal.isZero() == false && param.air_density) {
        f += area * param.air_density *
             air_damper::face_gradient(dt, y, x, normal, wind, param);
        H += area * param.air_density *
             air_damper::face_hessian(dt, normal, param);
    }

    bool pulled(false);
    for (unsigned j = 0; j < data.constraint.pull.size; ++j) {
        if (i == data.constraint.pull[j].index) {
            Vec3f position = data.constraint.pull[j].position;
            float weight = data.constraint.pull[j].weight;
            f += weight * (y - position);
            H += weight * Mat3x3f::Identity();
            pulled = true;
            break;
        }
    }

    if (!pulled) {
        f += mass * momentum::gradient(dt, y, target[i]);
        H += mass * momentum::hessian(dt);
    }

    if (param.isotropic_air_friction) {
        f += param.isotropic_air_friction * (y - x) / (dt * dt);
        H += (param.isotropic_air_friction / (dt * dt)) * Mat3x3f::Identity();
    }

    if (param.fix_xz && y[1] > float(param.fix_xz)) {
        float t = fmin(1.0f, y[1] - float(param.fix_xz));
        Vec3f n(0.0f, 1.0f, 0.0f);
        Mat3x3f P = Mat3x3f::Identity() - n * n.transpose();
        f += P * t * mass * (y - x) / (dt * dt);
        H += P * t * mass * (dt * dt);
    }

    Map<Vec3f>(force.data + 3 * i) += f;
    diag_hess[i] += H;
}

__device__ void embed_rod_force_hessian(const DataSet &data,
                                        const Vec<Vec3f> &eval_x,
                                        Vec<float> &force, FixedCSRMat &hess,
                                        float dt, const ParamSet &param,
                                        unsigned i) {
    const Vec2u &edge = data.mesh.mesh.edge[i];
    const Vec3f &x0 = eval_x[edge[0]];
    const Vec3f &x1 = eval_x[edge[1]];

    float l0 = data.prop.edge[i].length;
    Vec3f t = (x1 - x0);
    float l = t.norm();
    float mass = data.prop.edge[i].mass;
    float stiffness = data.prop.edge[i].stiffness;
    if (stiffness > 0.0f) {
        Mat3x2f dedx;
        Mat6x6f d2edx2;
        hook::make_diff_table(x0, x1, l0, stiffness * mass, dedx, d2edx2);
        utility::atomic_embed_force<2>(edge, dedx, force);
        utility::atomic_embed_hessian<2>(edge, d2edx2, hess);
    }
}

__device__ void embed_face_force_hessian(const DataSet &data,
                                         const Vec<Vec3f> &eval_x,
                                         Vec<float> &force, FixedCSRMat &hess,
                                         float dt, const ParamSet &param,
                                         unsigned i) {
    const Vec3u &face = data.mesh.mesh.face[i];
    const FaceProp &prop = data.prop.face[i];
    const Vec3f &x0 = eval_x[face[0]];
    const Vec3f &x1 = eval_x[face[1]];
    const Vec3f &x2 = eval_x[face[2]];
    Mat3x3f dedx = Mat3x3f::Zero();
    Mat9x9f d2edx2 = Mat9x9f::Zero();
    float mass = data.prop.face[i].mass;
    float mu = data.prop.face[i].mu;
    if (mu > 0.0f) {
        Mat3x2f F;
        Mat3x3f X;
        X << x0, x1, x2;
        F = utility::compute_deformation_grad(X, data.inv_rest2x2[i]);
        const Svd3x2 svd = utility::svd3x2(F);
        if (prop.model == Model::BaraffWitkin) {
            Mat3x2f de0dF = BaraffWitkin::stretch_gradient(F, mu);
            Mat3x2f de1dF = BaraffWitkin::shear_gradient(F, prop.lambda);
            Mat6x6f d2e0dF2 = BaraffWitkin::stretch_hessian(F, mu);
            Mat6x6f d2e1dF2 = BaraffWitkin::shear_hessian(F, prop.lambda);
            Mat3x2f dedF = de0dF + de1dF;
            Mat6x6f d2edF2 = d2e0dF2 + d2e1dF2;
            dedx += mass * utility::convert_force(dedF, data.inv_rest2x2[i]);
            d2edx2 +=
                mass * utility::convert_hessian(d2edF2, data.inv_rest2x2[i]);
        } else {
            DiffTable2 table;
            Mat3x2f dedF;
            Mat6x6f d2edF2;
            if (prop.model == Model::ARAP) {
                table = ARAP::make_diff_table2(svd.S, mu, prop.lambda);
            } else if (prop.model == Model::StVK) {
                table = StVK::make_diff_table2(svd.S, mu, prop.lambda);
            } else if (prop.model == Model::SNHk) {
                table = SNHk::make_diff_table2(svd.S, mu, prop.lambda);
            } else {
                assert(false);
            }
            dedF = eigenanalysis::compute_force(table, svd);
            d2edF2 = eigenanalysis::compute_hessian(table, svd,
                                                    param.eiganalysis_eps);
            dedx += mass * utility::convert_force(dedF, data.inv_rest2x2[i]);
            d2edx2 +=
                mass * utility::convert_hessian(d2edF2, data.inv_rest2x2[i]);
        }
        utility::atomic_embed_force<3>(face, dedx, force);
        utility::atomic_embed_hessian<3>(face, d2edx2, hess);
    }
}

__device__ void embed_tet_force_hessian(const DataSet &data,
                                        const Vec<Vec3f> &eval_x,
                                        Vec<float> &force, FixedCSRMat &hess,
                                        float dt, const ParamSet &param,
                                        unsigned i) {
    const Vec4u &tet = data.mesh.mesh.tet[i];
    const TetProp &prop = data.prop.tet[i];
    const float mass = prop.mass;
    const float mu = prop.mu;
    if (mu > 0.0f) {
        const float lambda = prop.lambda;
        const Vec3f &x0 = eval_x[tet[0]];
        const Vec3f &x1 = eval_x[tet[1]];
        const Vec3f &x2 = eval_x[tet[2]];
        const Vec3f &x3 = eval_x[tet[3]];
        Mat3x4f X;
        X << x0, x1, x2, x3;
        Mat3x3f F = utility::compute_deformation_grad(X, data.inv_rest3x3[i]);
        Svd3x3 svd = utility::svd3x3_rv(F);
        DiffTable3 table;
        Mat3x3f dedF;
        Mat9x9f d2edF2;
        Mat3x4f dedx = Mat3x4f::Zero();
        Mat12x12f d2edx2 = Mat12x12f::Zero();
        if (prop.model == Model::ARAP) {
            table = ARAP::make_diff_table3(svd.S, mu, lambda);
        } else if (prop.model == Model::StVK) {
            table = StVK::make_diff_table3(svd.S, mu, lambda);
        } else if (prop.model == Model::SNHk) {
            table = SNHk::make_diff_table3(svd.S, mu, lambda);
        } else {
            assert(false);
        }
        dedF = eigenanalysis::compute_force(table, svd);
        d2edF2 =
            eigenanalysis::compute_hessian(table, svd, param.eiganalysis_eps);
        dedx += mass * utility::convert_force(dedF, data.inv_rest3x3[i]);
        d2edx2 += mass * utility::convert_hessian(d2edF2, data.inv_rest3x3[i]);
        utility::atomic_embed_force<4>(tet, dedx, force);
        utility::atomic_embed_hessian<4>(tet, d2edx2, hess);
    }
}

__device__ void embed_hinge_force_hessian(const DataSet &data,
                                          const Vec<Vec3f> &eval_x,
                                          Vec<float> &force, FixedCSRMat &hess,
                                          const ParamSet &param, unsigned i) {
    const HingeProp &prop = data.prop.hinge[i];
    float length = prop.length;
    float bend = prop.bend;
    float ghat = prop.ghat;
    float stiff_k = 2.0f * bend * length * ghat;
    if (stiff_k > 0.0f) {
        Vec4u hinge = data.mesh.mesh.hinge[i];
        Mat3x4f dedx;
        Mat12x12f d2edx2;
        dihedral_angle::face_compute_force_hessian(eval_x, hinge, dedx, d2edx2);
        utility::atomic_embed_force<4>(hinge, stiff_k * dedx, force);
        utility::atomic_embed_hessian<4>(hinge, stiff_k * d2edx2, hess);
    }
}

__device__ void
embed_rod_bend_force_hessian(const DataSet &data, const Vec<Vec3f> &eval_x,
                             Vec<float> &force, FixedCSRMat &hess,
                             const ParamSet &param, unsigned i) {
    if (data.mesh.neighbor.vertex.edge.count(i) == 2 &&
        data.mesh.neighbor.vertex.face.count(i) == 0) {
        unsigned edge_idx_0 = data.mesh.neighbor.vertex.edge(i, 0);
        unsigned edge_idx_1 = data.mesh.neighbor.vertex.edge(i, 1);
        float bend_0 = data.prop.edge[edge_idx_0].bend;
        float bend_1 = data.prop.edge[edge_idx_1].bend;
        float bend = 0.5f * (bend_0 + bend_1);
        float mass = data.prop.vertex[i].mass;
        float stiff_k = bend * mass;
        if (mass > 0.0f && stiff_k > 0.0f) {
            Vec2u edge_0 = data.mesh.mesh.edge[edge_idx_0];
            Vec2u edge_1 = data.mesh.mesh.edge[edge_idx_1];
            unsigned j = edge_0[0] == i ? edge_0[1] : edge_0[0];
            unsigned k = edge_1[0] == i ? edge_1[1] : edge_1[0];
            Vec3u element = Vec3u(j, i, k);
            Vec3f x0 = eval_x[j];
            Vec3f x1 = eval_x[i];
            Vec3f x2 = eval_x[k];
            Mat3x3f dedx = dihedral_angle::strand_gradient(x0, x1, x2);
            Vec9f vec_dedx = Map<Vec9f>(dedx.data());
            Mat9x9f d2edx2 = vec_dedx * vec_dedx.transpose();
            utility::atomic_embed_force<3>(element, stiff_k * dedx, force);
            utility::atomic_embed_hessian<3>(element, stiff_k * d2edx2, hess);
        }
    }
}

void embed_momentum_force_hessian(const DataSet &data,
                                  const Vec<Vec3f> &eval_x,
                                  const Vec<Vec3f> &velocity, float dt,
                                  const Vec<Vec3f> &target, Vec<float> &force,
                                  Vec<Mat3x3f> &diag_hess,
                                  const ParamSet &param) {
    DISPATCH_START(data.vertex.curr.size)
    [data, eval_x, velocity, dt, target, force, diag_hess,
     param] __device__(unsigned i) mutable {
        if (data.prop.vertex[i].fix_index == 0) {
            energy::embed_vertex_force_hessian(data, eval_x, velocity, target,
                                               force, diag_hess, dt, param, i);
        }
    } DISPATCH_END;
}

void embed_elastic_force_hessian(const DataSet &data, const Vec<Vec3f> &eval_x,
                                 Vec<float> &force, FixedCSRMat &fixed_hess,
                                 float dt, const ParamSet &param) {
    unsigned surface_vert_count = data.surface_vert_count;
    unsigned hinge_count = data.mesh.mesh.hinge.size;
    unsigned shell_face_count = data.shell_face_count;
    unsigned rod_count = data.rod_count;
    unsigned tet_count = data.mesh.mesh.tet.size;

    DISPATCH_START(surface_vert_count)
    [data, eval_x, force, fixed_hess, dt,
     param] __device__(unsigned i) mutable {
        if (data.prop.vertex[i].fix_index == 0) {
            energy::embed_rod_bend_force_hessian(data, eval_x, force,
                                                 fixed_hess, param, i);
        }
    } DISPATCH_END;

    if (rod_count > 0) {
        DISPATCH_START(rod_count)
        [data, eval_x, force, fixed_hess, dt,
         param] __device__(unsigned i) mutable {
            if (!data.prop.edge[i].fixed) {
                energy::embed_rod_force_hessian(data, eval_x, force, fixed_hess,
                                                dt, param, i);
            }
        } DISPATCH_END;
    }

    if (shell_face_count > 0) {
        DISPATCH_START(shell_face_count)
        [data, eval_x, force, fixed_hess, dt,
         param] __device__(unsigned i) mutable {
            if (!data.prop.face[i].fixed) {
                energy::embed_face_force_hessian(data, eval_x, force,
                                                 fixed_hess, dt, param, i);
            }
        } DISPATCH_END;
    }

    if (tet_count > 0) {
        DISPATCH_START(tet_count)
        [data, eval_x, force, fixed_hess, dt,
         param] __device__(unsigned i) mutable {
            if (!data.prop.tet[i].fixed) {
                energy::embed_tet_force_hessian(data, eval_x, force, fixed_hess,
                                                dt, param, i);
            }
        } DISPATCH_END;
    }

    if (hinge_count > 0) {
        DISPATCH_START(hinge_count)
        [data, eval_x, force, fixed_hess,
         param] __device__(unsigned i) mutable {
            if (data.prop.hinge[i].fixed == false &&
                (data.mesh.type.hinge[i] & 1) == 0) {
                energy::embed_hinge_force_hessian(data, eval_x, force,
                                                  fixed_hess, param, i);
            }
        } DISPATCH_END;
    }
}

void embed_stitch_force_hessian(const DataSet &data, const Vec<Vec3f> &eval_x,
                                Vec<float> &force, FixedCSRMat &fixed_out,
                                const ParamSet &param) {
    unsigned seam_count = data.constraint.stitch.size;
    if (seam_count) {
        DISPATCH_START(seam_count)
        [data, eval_x, force, fixed_out, param] __device__(unsigned i) mutable {
            const Stitch &stitch = data.constraint.stitch[i];
            Vec3u index(stitch.index[0], stitch.index[1], stitch.index[2]);
            const Vec3f &x0 = eval_x[index[0]];
            const Vec3f &x1 = eval_x[index[1]];
            const Vec3f &x2 = eval_x[index[2]];
            float ghat_0 = data.prop.vertex[index[0]].ghat;
            float ghat_1 = data.prop.vertex[index[1]].ghat;
            float ghat_2 = data.prop.vertex[index[2]].ghat;
            float w[] = {1.0f, 1.0f - stitch.weight, stitch.weight};
            float l0 = (w[0] * ghat_0 + w[1] * ghat_1 + w[2] * ghat_2) / 2.0f;
            float s(1.0f / 3.0f);
            const Vec3f cog = s * x0 + s * x1 + s * x2;
            Vec3f z0 = w[0] * (x0 - cog);
            Vec3f z1 = w[1] * (x1 - cog) +
                       w[2] * (x2 - cog);
            Vec3f t = z0 - z1;
            float l = fmin(0.01f, t.norm());
            Vec3f n = t / l;
            Mat3x9f dtdx;
            dtdx << w[0] * Mat3x3f::Identity(), -w[1] * Mat3x3f::Identity(),
                -w[2] * Mat3x3f::Identity();
            Vec3f dedt = (l / l0 - 1.0f) * n;
            Vec9f g = dtdx.transpose() * n;
            float r = (l - l0) / l;
            float c0 = fmaxf(0.0f, 1.0f - r) / l0;
            float c1 = fmaxf(0.0f, r / l0);
            Mat3x3f gradient;
            gradient.col(0) = w[0] * dedt;
            gradient.col(1) = -w[1] * dedt;
            gradient.col(2) = -w[2] * dedt;
            Mat9x9f hessian =
                c0 * g * g.transpose() + c1 * dtdx.transpose() * dtdx;
            utility::atomic_embed_force<3>(
                index, param.stitch_stiffness * gradient, force);
            utility::atomic_embed_hessian<3>(
                index, param.stitch_stiffness * hessian, fixed_out);
        } DISPATCH_END;
    }
}

} // namespace energy
