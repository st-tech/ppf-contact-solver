#include "hip/hip_runtime.h"
#include "../common.hpp"

template <typename Lambda> __global__ void launch_kernel(Lambda func, int n) {
    unsigned idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        func(idx);
    }
}

#define DISPATCH_START(n)                                                      \
    {                                                                          \
        const unsigned n_threads(n);                                           \
        if (n_threads > 0) {                                                   \
            unsigned block_size;                                               \
            unsigned grid_size;                                                \
            if (n_threads < BLOCK_SIZE) {                                      \
                block_size = n_threads;                                        \
                grid_size = 1;                                                 \
            } else {                                                           \
                block_size = BLOCK_SIZE;                                       \
                grid_size = (n_threads + block_size - 1) / block_size;         \
            }                                                                  \
            launch_kernel<<<grid_size, block_size>>>(

#define DISPATCH_END , n_threads);                                             \
    hipError_t error = hipGetLastError();                                    \
    if (error != hipSuccess) {                                                \
        std::cerr << "CUDA error in file '" << __FILE__ << "' in line "        \
                  << __LINE__ << ": " << hipGetErrorString(error)             \
                  << std::endl;                                                \
        exit(1);                                                               \
    }                                                                          \
    }                                                                          \
    }
