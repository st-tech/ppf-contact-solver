#include "hip/hip_runtime.h"
// File: utility.cu
// Author: Ryoichi Ando (ryoichi.ando@zozo.com)
// License: Apache v2.0

#ifndef UTIL_HPP
#define UTIL_HPP

#include "dispatcher.hpp"
#include "utility.hpp"
#include <limits>

#define _real_ float
#define USE_EIGEN_SYMM_EIGSOLVE
#include "eig-hpp/eigsolve2x2.hpp"
#include "eig-hpp/eigsolve3x3.hpp"

namespace utility {

struct ReduceInfo {
    unsigned n = 0;
    unsigned *d_block_sums = nullptr;
    unsigned *h_results = nullptr;

    void init(unsigned n) {
        this->n = n;
        unsigned num_blocks = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
        CUDA_HANDLE_ERROR(
            hipMalloc(&d_block_sums, num_blocks * sizeof(unsigned)));
        h_results = new unsigned[num_blocks];
    }
};

static ReduceInfo reduce_info;

__device__ Vec3f compute_vertex_normal(const DataSet &data,
                                       const Vec<Vec3f> &vertex, unsigned i) {
    Vec3f normal = Vec3f::Zero();
    if (data.mesh.neighbor.vertex.face.size) {
        for (unsigned j = 0; j < data.mesh.neighbor.vertex.face.count(i); ++j) {
            const Vec3u &face =
                data.mesh.mesh.face[data.mesh.neighbor.vertex.face(i, j)];
            const Vec3f &z0 = vertex[face[0]];
            const Vec3f &z1 = vertex[face[1]];
            const Vec3f &z2 = vertex[face[2]];
            normal += (z1 - z0).cross(z2 - z0);
        }
        if (normal.squaredNorm()) {
            normal.normalize();
        }
    }
    return normal;
}

__device__ void solve_symm_eigen2x2(const Mat2x2f &matrix, Vec2f &eigenvalues,
                                    Mat2x2f &eigenvectors) {
    eig_tuple_2x2 result = sym_eigsolve_2x2(matrix);
    eigenvalues = result.lambda;
    eigenvectors = result.eigvecs;
}

__device__ void solve_symm_eigen3x3(const Mat3x3f &matrix, Vec3f &eigenvalues,
                                    Mat3x3f &eigenvectors) {
    eig_tuple_3x3 result = sym_eigsolve_3x3(matrix);
    eigenvalues = result.lambda;
    eigenvectors = result.eigvecs;
}

__device__ Svd3x2 svd3x2_shifted(const Mat3x2f &F) {
    Mat2x2f A = F.transpose() * F - Mat2x2f::Identity();
    eig_tuple_2x2 result = sym_eigsolve_2x2(A);
    Mat2x2f V = result.eigvecs;
    Mat3x2f U = F * V;
    for (int i = 0; i < U.cols(); i++) {
        U.col(i).normalize();
    }
    return {U, singular_vals_minus_one(F), V.transpose()};
}

__device__ Svd3x2 svd3x2(const Mat3x2f &F) {
    eig_tuple_2x2 result = sym_eigsolve_2x2(F.transpose() * F);
    Vec2f sigma = result.lambda;
    Mat2x2f V = result.eigvecs;
    for (int i = 0; i < 2; ++i) {
        sigma[i] = sqrtf(fmax(0.0f, sigma[i]));
    }
    Mat3x2f U = F * V;
    for (int i = 0; i < U.cols(); i++) {
        U.col(i).normalize();
    }
    return {U, sigma, V.transpose()};
}

__device__ Svd3x3 svd3x3(const Mat3x3f &F) {
    eig_tuple_3x3 result = sym_eigsolve_3x3(F.transpose() * F);
    Vec3f sigma = result.lambda;
    Mat3x3f V = result.eigvecs;
    for (int i = 0; i < 3; ++i) {
        sigma[i] = sqrtf(fmax(0.0f, sigma[i]));
    }
    Mat3x3f U = F * V;
    for (int i = 0; i < U.cols(); i++) {
        U.col(i).normalize();
    }
    return {U, sigma, V.transpose()};
}

__device__ Svd3x3 svd3x3_rv(const Mat3x3f &F) {
    Svd3x3 svd = svd3x3(F);
    float det_u = svd.U.determinant();
    float det_vt = svd.Vt.determinant();
    Mat3x3f L = Mat3x3f::Identity();
    unsigned min_index;
    svd.S.minCoeff(&min_index);
    L(min_index, min_index) = -1.0f;
    if (det_u < 0.0f && det_vt > 0.0f) {
        svd.U = svd.U * L;
        svd.S[min_index] *= -1.0f;
    } else if (det_u > 0.0f && det_vt < 0.0f) {
        svd.Vt = L * svd.Vt;
        svd.S[min_index] *= -1.0f;
    }
    return svd;
}

template <typename T> __device__ SMat<T, 3, 2> make_diff_mat3x2() {
    SMat<T, 3, 2> result = SMat<T, 3, 2>::Zero();
    result(0, 0) = T(-1.0f);
    result(0, 1) = T(-1.0f);
    result(1, 0) = T(1.0f);
    result(2, 1) = T(1.0f);
    return result;
}

template <typename T> __device__ SMat<T, 4, 3> make_diff_mat4x3() {
    SMat<T, 4, 3> result = SMat<T, 4, 3>::Zero();
    result(0, 0) = T(-1.0f);
    result(0, 1) = T(-1.0f);
    result(0, 2) = T(-1.0f);
    result(1, 0) = T(1.0f);
    result(2, 1) = T(1.0f);
    result(3, 2) = T(1.0f);
    return result;
}

__device__ Mat3x3f convert_force(const Mat3x2f &dedF,
                                 const Mat2x2f &inv_rest2x2) {
    const Mat3x2f g = make_diff_mat3x2<float>() * inv_rest2x2;
    Mat3x3f result;
    for (unsigned i = 0; i < 3; ++i) {
        for (unsigned dim = 0; dim < 3; ++dim) {
            result(dim, i) = g.row(i).dot(dedF.row(dim));
        }
    }
    return result;
}

__device__ Mat3x4f convert_force(const Mat3x3f &dedF,
                                 const Mat3x3f &inv_rest3x3) {
    const Mat4x3f g = make_diff_mat4x3<float>() * inv_rest3x3;
    Mat3x4f result;
    for (unsigned i = 0; i < 4; ++i) {
        for (unsigned dim = 0; dim < 3; ++dim) {
            result(dim, i) = g.row(i).dot(dedF.row(dim));
        }
    }
    return result;
}

__device__ Mat9x9f convert_hessian(const Mat6x6f &d2ed2f,
                                   const Mat2x2f &inv_rest2x2) {
    const Mat3x2f g = make_diff_mat3x2<float>() * inv_rest2x2;
    Mat6x9f dfdx;
    for (unsigned j = 0; j < 9; ++j) {
        Mat3x3f dx_mat = Mat3x3f::Zero();
        Map<Vec9f>(dx_mat.data())[j] = 1.0f;
        Mat3x2f tmp = dx_mat * g;
        dfdx.col(j) = Map<Vec6f>(tmp.data());
    }
    Mat9x9f result = Mat9x9f::Zero();
    for (unsigned i = 0; i < 6; ++i) {
        for (unsigned j = 0; j < 6; ++j) {
            result += d2ed2f(i, j) * dfdx.row(i).transpose() * dfdx.row(j);
        }
    }
    return result; // dfdx.transpose() * d2ed2f * dfdx;
}

__device__ Mat12x12f convert_hessian(const Mat9x9f &d2ed2f,
                                     const Mat3x3f &inv_rest3x3) {
    const Mat4x3f g = make_diff_mat4x3<float>() * inv_rest3x3;
    Mat9x12f dfdx;
    for (unsigned j = 0; j < 12; ++j) {
        Mat3x4f dx_mat = Mat3x4f::Zero();
        Map<Vec12f>(dx_mat.data())[j] = 1.0f;
        Mat3x3f tmp = dx_mat * g;
        dfdx.col(j) = Map<Vec9f>(tmp.data());
    }
    Mat12x12f result = Mat12x12f::Zero();
    for (unsigned i = 0; i < 9; ++i) {
        for (unsigned j = 0; j < 9; ++j) {
            result += d2ed2f(i, j) * dfdx.row(i).transpose() * dfdx.row(j);
        }
    }
    return result; // dfdx.transpose() * d2ed2f * dfdx;
}

__device__ Mat3x2f compute_deformation_grad(const Mat3x3f &x,
                                            const Mat2x2f &inv_rest2x2) {
    return (x * make_diff_mat3x2<float>()) * inv_rest2x2;
}

__device__ Mat3x3f compute_deformation_grad(const Mat3x4f &x,
                                            const Mat3x3f &inv_rest3x3) {
    return (x * make_diff_mat4x3<float>()) * inv_rest3x3;
}

__device__ float compute_face_area(const Mat3x3f &vertex) {
    const Vec3f v0 = vertex.col(0);
    const Vec3f v1 = vertex.col(1);
    const Vec3f v2 = vertex.col(2);
    return 0.5f * (v1 - v0).cross(v2 - v0).norm();
}

template <class T, class Y, typename Op>
__global__ void reduce_op_kernel(const T *input, Y *output, Op func, Y init_val,
                                 unsigned n) {
    __shared__ Y shared_data[BLOCK_SIZE];
    unsigned tid = threadIdx.x;
    unsigned global_idx = blockIdx.x * blockDim.x + tid;
    shared_data[tid] = (global_idx < n) ? input[global_idx] : init_val;
    __syncthreads();
    for (unsigned stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (tid < stride) {
            shared_data[tid] =
                func(shared_data[tid], shared_data[tid + stride]);
        }
        __syncthreads();
    }
    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}

template <class T, class Y, typename Op>
Y reduce(const T *d_input, Op func, Y init_val, unsigned n) {
    if (n > 0) {
        if (sizeof(Y) * n <= sizeof(unsigned) * reduce_info.n) {
            unsigned grid_size = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
            Y *d_output = reinterpret_cast<Y *>(reduce_info.d_block_sums);
            Y *h_results = reinterpret_cast<Y *>(reduce_info.h_results);
            size_t shared_mem_size = sizeof(Y) * BLOCK_SIZE;
            reduce_op_kernel<T, Y><<<grid_size, BLOCK_SIZE, shared_mem_size>>>(
                d_input, d_output, func, init_val, n);
            hipMemcpy(h_results, d_output, grid_size * sizeof(Y),
                       hipMemcpyDeviceToHost);
            Y result = init_val;
            for (unsigned i = 0; i < grid_size; i++) {
                result = func(result, h_results[i]);
            }
            return result;
        } else {
            fprintf(stderr, "Error: reduce buffer size is too small\n");
            fprintf(stderr, "n: %u, reduce_info.n: %u\n", n, reduce_info.n);
            exit(1);
        }
    } else {
        return init_val;
    }
}

template <class T> T sum_array(Vec<T> array, unsigned size) {
    return reduce<T, T>(
        array.data, [] __host__ __device__(T a, T b) { return a + b; }, T(),
        size);
}

template <class T> unsigned sum_integer_array(Vec<T> array, unsigned size) {
    return reduce<T, unsigned>(
        array.data, [] __host__ __device__(T a, T b) { return a + b; }, 0u,
        size);
}

template <class T> T min_array(const T *array, unsigned size, T init_val) {
    return reduce<T, T>(
        array, [] __host__ __device__(T a, T b) { return a < b ? a : b; },
        init_val, size);
}

template <class T> T max_array(const T *array, unsigned size, T init_val) {
    return reduce<T, T>(
        array, [] __host__ __device__(T a, T b) { return a > b ? a : b; },
        init_val, size);
}

void compute_svd(DataSet data, Vec<Vec3f> curr, Vec<Svd3x2> svd,
                 ParamSet param) {
    unsigned shell_face_count = data.shell_face_count;
    DISPATCH_START(shell_face_count)
    [data, curr, svd, param] __device__(unsigned i) mutable {
        Vec3u face = data.mesh.mesh.face[i];
        Mat3x3f x;
        x << curr[face[0]], curr[face[1]], curr[face[2]];
        const Mat3x2f F =
            utility::compute_deformation_grad(x, data.inv_rest2x2[i]);
        svd[i] = utility::svd3x2(F);
    } DISPATCH_END;
}

__device__ float get_wind_weight(float time) {
    float angle = 30.0f * time;
    float t = 0.25f;
    return t * (0.5f * (1.0f + sinf(angle))) + (1.0f - t);
}

void set_max_reduce_count(unsigned n) { reduce_info.init(n); }

} // namespace utility

template float utility::sum_array(Vec<float> array, unsigned size);
template unsigned utility::sum_integer_array(Vec<unsigned> array,
                                             unsigned size);
template unsigned utility::sum_integer_array(Vec<char> array, unsigned size);
template float utility::min_array(const float *array, unsigned size,
                                  float init_val);
template float utility::max_array(const float *array, unsigned size,
                                  float init_val);
template char utility::min_array(const char *array, unsigned size,
                                 char init_val);
template char utility::max_array(const char *array, unsigned size,
                                 char init_val);
template unsigned utility::min_array(const unsigned *array, unsigned size,
                                     unsigned init_val);
template unsigned utility::max_array(const unsigned *array, unsigned size,
                                     unsigned init_val);

#endif
